#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../libs/mmio.c"
#include <sys/time.h>


//#define int unsigned int



struct COO{
    int* rows;
    int* cols;
    int nodes;
    int edges;
};

struct CSR{
    int* pointers;
    int* indices;
    int nodes;
    int nz;

};

__host__ struct COO create_coo(struct COO coo, int* I, int* J, int M, int nz);
__host__ struct CSR coo_to_csr(struct COO coo);

__host__ void ElapsedTime(timeval t1, timeval t2);

__global__ void Calc_p1(int* pointers, int* p1, int N){
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        p1[i] = pointers[i+1] - pointers[i]; 
    }
}

__global__ void Calc_d3(int* p1, int* d3, int* c3, int N){
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        d3[i] = p1[i]*(p1[i] - 1) / 2;
        if(c3[i])
            d3[i] -= c3[i];
    }
}

__global__ void Calc_p2(int* p1, int* p2, int* c3, int* pointers, int* indices, int N){
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        int start = pointers[i], end = pointers[i+1];
        for(int j = start; j < end; j++){
            p2[i] += p1[indices[j]];
        }
        p2[i] -= p1[i];
        if(c3[i])
            p2[i] -= 2*c3[i];
    }
}

__global__ void Calc_c3(int* c3, int* pointers, int* indices, int N){
    int i =  blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        int start = pointers[i], end = pointers[i+1];
        int count = 0;
        int temp_j, temp_k, temp_index, temp_start, temp_end;
        for(int j = start; j < end; j++){
            temp_index = indices[j];
            temp_start = pointers[temp_index];
            temp_end = pointers[temp_index + 1];
            temp_j = start;
            temp_k = temp_start;
            while(temp_j < end && temp_k < temp_end){
                if(indices[temp_j] == indices[temp_k]){
                    temp_j++;
                    temp_k++;
                    count++;
                }
                else if(indices[temp_j] < indices[temp_k]){
                    temp_j++;
                }
                else{
                    temp_k++;
                }
            }
        }
        c3[i] = (int) (count / 2);
        
    }

}



int main(int argc, char* argv[]){

    // mtx read 
    timeval start_time, read_time, time, transform_time;  //measure elapsed time
    gettimeofday(&start_time, NULL);
    int ret_code;
    MM_typecode matcode;
    FILE *f;
    int M, N, nz;   
    int i, *I, *J;
    double *val;

    if (argc < 2)
	{
		fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
		exit(1);
	}
    else    
    { 
        if ((f = fopen(argv[1], "r")) == NULL) 
            exit(1);
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) && 
            mm_is_sparse(matcode) )
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    // find out size of sparse matrix

    if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) !=0)
        exit(1);


    // reseve memory for matrices 
    I = (int *) malloc(nz * sizeof(int));
    J = (int *) malloc(nz * sizeof(int));


    for (i=0; i<nz; i++)
    {
        fscanf(f, "%d %d\n", &I[i], &J[i]);
        I[i]--;  // adjust from 1-based to 0-based 
        J[i]--;
    }

    //close file
    if (f !=stdin) fclose(f);
    
    
    //mtx file banner and sizes
    mm_write_banner(stdout, matcode);
    mm_write_mtx_crd_size(stdout, M, N, nz);

    gettimeofday(&read_time, NULL);
    printf("read time: ");
    ElapsedTime(start_time, read_time);


    struct COO coo = create_coo(coo, I, J, M, nz);
    struct CSR csr = coo_to_csr(coo);

    //csr format created, free space from coo    
    free(I);
    free(J);
    free(coo.cols);
    free(coo.rows);

    gettimeofday(&transform_time, NULL);
    printf("transform time: ");
    ElapsedTime(read_time, transform_time);



    int* d_pointers, *d_indices, *d_p1, *d_p2, *d_d3, *d_c3;    //device
    int *h_pointers, *h_indices, *h_p1, *h_p2, *h_d3, *h_c3;    //host
    int threadsPerBlock = 1024;
    int Blocks = (N + threadsPerBlock - 1)/threadsPerBlock;

    //allocate memory on host for results
    h_p1 = (int*) malloc(csr.nodes * sizeof(int));  
    h_p2 = (int*)malloc(csr.nodes * sizeof(int));
    h_d3 = (int*)malloc(csr.nodes * sizeof(int));
    h_c3 = (int*)malloc(csr.nodes * sizeof(int));
    h_pointers = csr.pointers;
    h_indices = csr.indices;
    
    
    hipMalloc(&d_pointers, (csr.nodes + 1) * sizeof(int));
    hipMemcpy(d_pointers, h_pointers, (csr.nodes+1) * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&d_p1, csr.nodes * sizeof(int));
    Calc_p1<<<Blocks, threadsPerBlock>>>(d_pointers, d_p1, csr.nodes);
    hipMemcpy(h_p1, d_p1, csr.nodes * sizeof(int), hipMemcpyDeviceToHost);

    hipMalloc(&d_indices, csr.nz * sizeof(int));
    hipMemcpy(d_indices, h_indices, csr.nz * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&d_c3, csr.nodes * sizeof(int));
    Calc_c3<<<Blocks, threadsPerBlock>>>(d_c3, d_pointers, d_indices, csr.nodes);
    hipMemcpy(h_c3, d_c3, csr.nodes * sizeof(int), hipMemcpyDeviceToHost);
    
    hipMalloc(&d_d3, csr.nodes * sizeof(int));
    Calc_d3<<<Blocks, threadsPerBlock>>>(d_p1, d_d3, d_c3, csr.nodes);
    hipMemcpy(h_d3, d_d3, csr.nodes * sizeof(int), hipMemcpyDeviceToHost);

    hipMalloc(&d_p2, csr.nodes * sizeof(int));
    Calc_p2<<<Blocks, threadsPerBlock>>>(d_p1, d_p2, d_c3, d_pointers, d_indices, csr.nodes);
    hipMemcpy(h_p2, d_p2, csr.nodes * sizeof(int), hipMemcpyDeviceToHost);


    
    //free memory on device
    hipFree(d_pointers);
    hipFree(d_indices);
    hipFree(d_c3);
    hipFree(d_d3);
    hipFree(d_p1);
    hipFree(d_p2);
    
    //free memory on host
    free(csr.pointers);
    free(h_c3);
    free(h_d3);
    free(h_p1);
    free(h_p2);
    free(csr.indices);

    gettimeofday(&time, NULL);
    printf("exe time: ");
    ElapsedTime(transform_time, time);
    
    printf("Time: ");
    ElapsedTime(start_time, time);
    
    return 0;
}




void ElapsedTime(timeval t1, timeval t2){
    double time;
    time = t2.tv_sec - t1.tv_sec;
    time += (t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("%lf sec.\n", time);
}

struct CSR coo_to_csr(struct COO coo){

    struct CSR csr;
    csr.nodes = coo.nodes;
    csr.nz = coo.edges;
    csr.pointers = (int*)malloc((csr.nodes + 1) * sizeof(int));
    csr.indices = (int*)malloc(csr.nz * sizeof(int));
    
    for(int i = 0; i < csr.nodes; i++)
        csr.pointers[i] = 0;

    for(int i = 0; i < csr.nz; i++)
        csr.indices[i] = 0;

    for(int i = 0; i < csr.nz; i++)
        csr.pointers[coo.rows[i]]++;

    int temp = 0;
    int sum = 0;
    for(int i = 0; i < csr.nodes; i++){
        temp = csr.pointers[i];
        csr.pointers[i] = sum;
        sum += temp;
    }
    csr.pointers[csr.nodes] = csr.nz;

    int row = 0;
    int dest = 0;
    
    for(int i = 0; i < csr.nz; i++){
        row = coo.rows[i];
        dest = csr.pointers[row];
        csr.indices[dest] = coo.cols[i];
        csr.pointers[row]++;
    }
    
    temp = 0;
    int last = 0;
    
    for(int i = 0; i < csr.nodes + 1; i++){
        temp = csr.pointers[i];
        csr.pointers[i] = last;
        last = temp;
    }

    return csr;
}
struct COO create_coo(struct COO coo, int* I, int* J, int M, int nz){

    coo.nodes = M;
    coo.edges = 2 * nz;
    coo.cols = (int*)calloc(coo.edges, sizeof(int));
    coo.rows = (int*)calloc(coo.edges, sizeof(int));
    for(int i = 0; i < coo.edges; i++){
        if(i < nz){
            coo.cols[i] = J[i];
            coo.rows[i] = I[i];
        }
        else{
            coo.cols[i] = I[i - nz];
            coo.rows[i] = J[i - nz];
        }
    }
    
    return coo;
}


